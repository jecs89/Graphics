
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <random>
#include <iomanip>
#include <string>
#include <sstream>
#include <fstream>

//./fish 20000 10 200 15 5 

#define PI 3.1416

using namespace std;

template <typename T>
struct point{
	T x;
	T y;
	T z;
};

typedef point<double> p3D;

template <typename T>
struct fish{
	int num;
	T c;	//position
	T v;	//direction
	double s;	//speed
	double theta;	//vision's angle
	double r_r;		//repulsion radius
	double r_p;		//reach radius
	double w_a;		//attraction weight
	double w_o;		//orientation weight

	vector<int> neighborhood_r;	//neighbors in repulsion zone
	vector<int> neighborhood_p; //neighbors in annulus r_r & r_p
};

__global__ void cuda_norm( p3D* schoolfish, int pos, double* c){

	int index = threadIdx.x + blockIdx.x*blockDim.x;
	// printf( "%i\n", index );

	double t1 = powf(schoolfish[pos].x - schoolfish[index].x,2);
	double t2 = powf(schoolfish[pos].y - schoolfish[index].y,2);
	double t3 = powf(schoolfish[pos].z - schoolfish[index].z,2);
	double dist = sqrtf( t1 + t2 + t3 );

	// printf( "%i\n%.4f\t%.4f\t%.4f\n%.4f\t%.4f\t%.4f\n%.4f\t%.4f\t%.4f\t%.4f\n\n", index, schoolfish[pos].c.x, schoolfish[pos].c.y, schoolfish[pos].c.z, 
	// schoolfish[index].c.x, schoolfish[index].c.y, schoolfish[index].c.z, t1, t2, t3, dist);

	c[index] = dist ;
	// printf( "%d\n", c[index] );
}

template <typename Type>
void ptr2vector( Type*& ptr, vector< Type >& vec, int size){
	for( int i = 0; i < size; i++){
		// cout << "Start Ptr2Vec\n";
		vec.push_back( ptr[i] );
		// cout << "End Ptr2Vec\n";
	}
}

template <typename Type>
void vector2ptr( vector< Type >& vec, Type*& ptr, int size){
	// cout << "Start Vec2Ptr\n";
	for( int i = 0; i < size; i++){
		ptr[i] = vec[i];
	}
	// cout << "End Vec2Ptr\n";
}

// void ptrd2vectord( double*& ptr, vector< double >& vec, int size){
// 	cout << "Start PtrD2VecD\n";
// 	for( int i = 0; i < size; i++){
// 		cout << i << "\t" ;
// 		vec.push_back( ptr[i] );
// 	}
// 	cout << "End PtrD2VecD\n";
// }

// // template <typename Type>
// void vectord2ptrd( vector<double>& vec, double*& ptr, int size){
// 	cout << "Start VecD2PtrD\n";
// 	for( int i = 0; i < size; i++){
// 		cout << i << "\t" ;
// 		ptr[i] = vec[i];
// 	}
// 	cout << "End VecD2PtrD\n";
// }

template <typename T>
void printelem( T elem ){
	// cout << sizeof(elem) << endl;
	// cout << sizeof(elem[0]) << endl;
	if( sizeof(elem) == 1 || sizeof(elem) == 2 || sizeof(elem) == 4 ){
		// cout << "HOLA\n" ;
		// cout << elem << endl;
	}
	else if( sizeof(elem) > 4){
		if( sizeof(elem[0]) == 1 || sizeof(elem[0]) == 2 || sizeof(elem[0]) == 4){
			for( int i = 0; i < elem.size(); ++i){
				cout << elem[i] << "\t";
			}
			cout << endl;
		}
		 else if( sizeof(elem[0]) > 4){
		 	cout << elem.size() << endl;

			// for( int i = 0; i < elem.size(); ++i){
			// 	for( int j = 0; j < elem[0].size(); ++j){
			// 		cout << elem[i][j] << "\t";	
			// 	}
			// 	cout << endl;		
			// }
			// cout << endl;	
		}
	}
}

double calc_angle( p3D x, p3D y ){
	double den = x.x * y.x + x.y * y.y + x.z * y.z,
		  num = sqrtf( powf(x.x,2) + powf(x.y,2) + powf(x.z,2) ) + sqrtf( powf(y.x,2) + powf(y.y,2) + powf(y.z,2) );

	return acos( den / num );
}

void test_calc_angle(){
	p3D x, y;
	x.x = 2, x.y = 3, x.z = 0, y.x = 2, y.y = 0, y.z = 3;

    double ang = calc_angle( x, y );
    cout << setprecision(4) << ang << "\t" << ang*180/PI << endl;
}

double calc_norm( p3D x, p3D y){
	return sqrtf( powf(x.x-y.x,2) + powf( x.y-y.y,2) + powf( x.z-y.z,2) );
}

template <typename T>
class SchoolFish{
	vector< fish< T > > schoolfish;
	double lim1, lim2;

	vector<p3D> v_p3D;

	public:
		SchoolFish(int size){
		schoolfish.resize(size);
		v_p3D.resize(size);
		// double w_a, w_o;
	}

		void init( double wa, double wo ){

			int scale = 10;

			default_random_engine rng(random_device{}()); 		
			// uniform_real_distribution<double> dist( -10, 10); 

			lim1 = 50, lim2 = 100;
			uniform_real_distribution<double> dist( lim1, lim2 );

			for( int i = 0; i < schoolfish.size(); ++i){
				schoolfish[i].num = i;
				// schoolfish[i].c = pair<double,double> ( dist(rng),dist(rng) );
				// schoolfish[i].v = pair<double,double> (2,2);
				// schoolfish[i].v = pair<double,double> (dist(rng),dist(rng) );

				schoolfish[i].c.x = 1+i/2.0;
				schoolfish[i].c.y = 1+i/2.0;
				schoolfish[i].c.z = 1+i/2.0 ;

				schoolfish[i].v.x = dist(rng)/scale;
				schoolfish[i].v.y = dist(rng)/scale;
				schoolfish[i].v.z = dist(rng)/scale;

				schoolfish[i].s = 0.1;
				schoolfish[i].theta = PI/6;
				schoolfish[i].r_r = 0.5;
				schoolfish[i].r_p = 1;
				schoolfish[i].w_a = wa;
				schoolfish[i].w_o = wo;

				v_p3D[i] = schoolfish[i].c;
			}
		}

		void print(){
			int dec=4;
			int pos=dec*2;
			cout << setw(pos) << "# Fish" << setw(pos*3) << "C" << setw(pos*3) << "V" << setw(pos) << "S" << setw(pos) << "Theta" << setw(pos) << "R-R" << setw(pos)
				 << setw(pos) << "R-P" << setw(pos) << "W_A" << setw(pos) << "W_O" << endl;

			for( int i = 0; i < schoolfish.size(); ++i){
				cout << setprecision(dec) << setw(pos) << schoolfish[i].num << setw(pos) << schoolfish[i].c.x << setw(pos) << schoolfish[i].c.y << setw(pos) << schoolfish[i].c.z << setw(pos) << schoolfish[i].v.x
					 << setw(pos) << schoolfish[i].v.y << setw(pos) << schoolfish[i].v.z << setw(pos) << schoolfish[i].s << setw(pos) << schoolfish[i].theta 
					 << setw(pos) << schoolfish[i].r_r << setw(pos) << schoolfish[i].r_p << setw(pos) << schoolfish[i].w_a 
					 << setw(pos) << schoolfish[i].w_o << endl;
			}	
		}

		void print2file( ostream& file, int type ){
			int dec=4;
			int pos=dec*2;
			
			//file << setw(pos) << "# Fish" << setw(pos*2) << "C" << endl;
			for( int i = 0; i < schoolfish.size(); ++i){
				if( type == 1){
					file << setprecision(dec) << setw(pos) << schoolfish[i].num << setw(pos) << schoolfish[i].c.x << setw(pos) << schoolfish[i].c.y << setw(pos) << schoolfish[i].c.z << endl;
				}
				else if( type == 2){
					file << setprecision(dec) << schoolfish[i].c.x << "," <<  schoolfish[i].c.y << "," << schoolfish[i].c.z<< endl;	
				}
			}
		}

		vector<double> v_norm(int pos){

			// cout << "Start CNorm\n";

			int N = 10000, M = 1024;
			// cout << "Fish" << pos << endl;
			
			vector<double> ans;

			// ans[0] = 2.0;

			int size_ans = schoolfish.size() * sizeof(double);
			int size_school = v_p3D.size() * sizeof( p3D );

			// printf("%s\t%i\t%s\t%i\t%s\t%i\n", "School", (int)schoolfish.size(), "Size ans", size_ans, "Size school", size_school);

			double* ansptr;
			ansptr = (double *)malloc( size_ans );

			//Copy info from schoolfish to schoolptr
			p3D* schoolptr;
			schoolptr = ( p3D* )malloc( size_school );

			vector2ptr( v_p3D, schoolptr, schoolfish.size() );

			// cout << "Step\n";

			// cout << "Original \n";
			// this->print();

			// cout << "Copy of schoolfish\n";

			// int pos_ = 8;
			// int dec = 4;
			// for( int i = 0; i < schoolfish.size(); ++i){
			// 	cout << setprecision(dec) << setw(pos_) << schoolptr[i].num << setw(pos_) << schoolptr[i].c.x << setw(pos_) << schoolptr[i].c.y << setw(pos_) << schoolptr[i].c.z << setw(pos_) << schoolptr[i].v.x
			// 		 << setw(pos_) << schoolptr[i].v.y << setw(pos_) << schoolptr[i].v.z << setw(pos_) << schoolptr[i].s << setw(pos_) << schoolptr[i].theta 
			// 		 << setw(pos_) << schoolptr[i].r_r << setw(pos_) << schoolptr[i].r_p << setw(pos_) << schoolptr[i].w_a 
			// 		 << setw(pos_) << schoolptr[i].w_o << endl;
				
			// 	string blank = "      ";
			// 	// printf( "%s%.4f%s%.4f%s%.4f%s%.4f%s%.4f%s%.4f%s%.4f%s%.4f%s%.4f%s%.4f%s", blank.c_str(), schoolfish[i].c.x, blank.c_str(), schoolfish[i].c.second, 
			// 	// 		blank.c_str(), schoolfish[i].v.first, blank.c_str(), schoolfish[i].v.second, blank.c_str(), schoolfish[i].s, blank.c_str(), schoolfish[i].theta,
			// 	// 		blank.c_str(), schoolfish[i].r_r, blank.c_str(), schoolfish[i].r_p, blank.c_str(), schoolfish[i].w_a, blank.c_str(), schoolfish[i].w_o, "\n" );
			// }
			// cout << endl;	

			//Copy for devices
			p3D* d_schoolptr;
			double* d_ansptr;

			// // // Allocate space for device copies of schoolptr
			hipMalloc((void **)&d_schoolptr, size_school );
			hipMalloc((void **)&d_ansptr, size_ans );
			
			// // Copy inputs to device
			hipMemcpy(d_schoolptr, schoolptr, size_school, hipMemcpyHostToDevice);  // Args: Dir. destino, Dir. origen, tamano de dato, sentido del envio

			// // Launch add() kernel on GPU
			// cout << "Start Cuda Call\n";
			cuda_norm<<<(N+M-1)/M,M>>> ( d_schoolptr, pos, d_ansptr );
			// cout << "End Cuda Call\n";

			// // Copy result back to host
			hipMemcpy(ansptr, d_ansptr, size_ans, hipMemcpyDeviceToHost);

			ptr2vector( ansptr, ans, schoolfish.size() );

			// printelem( ans );

			// Cleanup
			free(schoolptr);
			free(ansptr);
			hipFree(d_ansptr);
			hipFree(d_schoolptr);

			// cout << "Finish CNorm\n";

			return ans;
		}

		void print_distances(){
			for( int i = 0; i < schoolfish.size(); ++i){
				printelem( v_norm(i) );
			}
		}

		void calc_neighboors( int k ){
			// cout << "Star neighbors\n";
			for( int i = 0; i < schoolfish.size(); ++i){
				vector<double> v_dist = ( v_norm(i) );

				for( int j = 0; j < v_dist.size(); ++j){
					int act_size = schoolfish[i].neighborhood_r.size()+ schoolfish[i].neighborhood_p.size();
					if( act_size < k ){
						if( v_dist[j] <= schoolfish[i].r_r ){
							if( schoolfish[i].num != j ){
								schoolfish[i].neighborhood_r.push_back( j );
								// cout << j << "\t";
							}
						}
						else if( v_dist[j] > schoolfish[i].r_r && v_dist[j] <= schoolfish[i].r_p ){
							if( schoolfish[i].num != j ){
								schoolfish[i].neighborhood_p.push_back( j );
								// cout << j << "\t";
							}
						}
					}
					else if( act_size == k ){
						break;
					}
				}
				// cout << endl;
			}
			// cout << "Good Calc neighbors" << endl;
		}

		void print_neighboors(){
			for( int i = 0; i < schoolfish.size(); ++i){
				cout << "Fish " << i << endl;
				
				cout << "Repulsion zone:" << schoolfish[i].neighborhood_r.size() << endl;
				printelem( schoolfish[i].neighborhood_r );

				cout << "P zone:" << schoolfish[i].neighborhood_p.size() << endl;
				printelem( schoolfish[i].neighborhood_p);
			}

		}

		void update_c(){
			for( int i = 0; i < schoolfish.size(); ++i){
				// pair<double,double> d(0,0);
				p3D d; d.x = d.y = d.z = 0;

				if( schoolfish[i].neighborhood_r.size() > 0 ){
					for( int j = 0; j < schoolfish[i].neighborhood_r.size(); ++j){
						int ii = schoolfish[i].num, jj = schoolfish[i].neighborhood_r[j];
						double den = calc_norm( schoolfish[ii].c, schoolfish[jj].c );
						d.x +=  ( schoolfish[jj].c.x - schoolfish[ii].c.x ) / ( den );
						d.y +=  ( schoolfish[jj].c.y - schoolfish[ii].c.y ) / ( den );	
						d.z += ( schoolfish[jj].c.z - schoolfish[ii].c.z ) / ( den );	
					}
					d.x = -1 * d.x; d.y = -1 * d.y; d.z = -1 * d.z;
				}
				else{
					for( int j = 0; j < schoolfish[i].neighborhood_r.size(); ++j){
						int ii = schoolfish[i].num, jj = schoolfish[i].neighborhood_r[j];
						double den = calc_norm( schoolfish[ii].c, schoolfish[jj].c );

						int norm_v = (schoolfish[i].v.x/ sqrtf( powf(schoolfish[i].v.x,2) + powf(schoolfish[i].v.y,2) + powf(schoolfish[i].v.z,2) ) );

						d.x +=  schoolfish[0].w_a * ( schoolfish[jj].c.x - schoolfish[ii].c.x ) / ( den ) + schoolfish[0].w_o * norm_v;
						d.y +=  schoolfish[0].w_a * ( schoolfish[jj].c.y - schoolfish[ii].c.y ) / ( den ) + schoolfish[0].w_o * norm_v;	
						d.z +=  schoolfish[0].w_a * ( schoolfish[jj].c.z - schoolfish[ii].c.z ) / ( den ) + schoolfish[0].w_o * norm_v;	
					}
					d.x = -1 * d.x; d.y = -1 * d.y; d.z = -1 * d.z;

				}

				if( calc_angle( schoolfish[i].c, d ) <= schoolfish[i].theta ){
					 schoolfish[i].c.x = schoolfish[i].c.x + schoolfish[i].s * d.x;
					 schoolfish[i].c.y = schoolfish[i].c.y + schoolfish[i].s * d.x;
					 schoolfish[i].c.z = schoolfish[i].c.z + schoolfish[i].s * d.x;
				}
			}
		}

		void movement( double t ){
			for( int i = 0; i < schoolfish.size(); ++i){
				// double cx = schoolfish[i].c.x, cy = schoolfish[i].c.second;

				check_limits( schoolfish[i].c, schoolfish[i].v ) ;

				schoolfish[i].c.x += schoolfish[i].v.x * schoolfish[i].s * t;
				schoolfish[i].c.y += schoolfish[i].v.y * schoolfish[i].s * t;
				schoolfish[i].c.z += schoolfish[i].v.z * schoolfish[i].s * t;
			}
			// cout << "Good Movement" << endl;
		}

		void check_limits( p3D& p, p3D& v ){

			int lmin = -100, lmax = 100;

			if( p.x + v.x < lmin || p.x + v.x > lmax ){
				v.x= v.x* -1;
			}
			else if( p.y + v.y < lmin || p.y + v.y > lmax){
				v.y = v.y * -1;
			}
			else if( p.z + v.z < lmin || p.z + v.z > lmax){
				v.z = v.z * -1;
			}
		}
};

template <typename T>
inline void str2num(string str, T& num){
	if ( ! (istringstream(str) >> num) ) num = 0;
}

// void test_vector2ptr(){
// 	int* a;
// 	vector<int> b;
// 	b.push_back(-10);
// 	vector2ptr(b,a,b.size()); 

// 	cout << a[0] << endl;
// }

// void test_ptr2vector(){
// 	int* ptr = ( int* )malloc( 1 * sizeof(int) );
// 	vector<int> a;
// 	ptr[0] = 100;

// 	ptr2vector(ptr,a,1);
// 	cout << a[0] << endl;
// }

int main( int argc, char** argv ){

	time_t timer = time(0); 

	//Params of SchoolFish
	int num_fish, k, iter;
	string par = argv[1]; str2num( par, num_fish);
	par = argv[2]; str2num( par, k);

	double t = 1;
	par = argv[3]; str2num( par, iter);

	double wa, wo;
	par = argv[4]; str2num( par, wa);
	par = argv[5]; str2num( par, wo);

	//Initialization of values
	// SchoolFish<p2D> myschool( num_fish );
	SchoolFish<p3D> myschool( num_fish );
	myschool.init(wa,wo);
	// myschool.print();

    ofstream result("movement_cuda.data");

	for( int i = 0; i < iter; ++i){
		// cout << "Iter: " << i << endl;
		// myschool.print_distances();
		myschool.movement(t);
		myschool.calc_neighboors(k);
		// myschool.print_neighboors();
		myschool.update_c();
		// myschool.print();
		myschool.print2file( result, 2);
	}

	result.close();	

	time_t timer2 = time(0); 
    cout <<"\nTiempo total: " << difftime(timer2, timer) << endl;

	return 0;
}